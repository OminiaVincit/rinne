#include "hip/hip_runtime.h"
#include "rinne.hpp"

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <unistd.h>

#ifdef __APPLE__
  #include <GLUT/glut.h>
#else
  #include <GL/glut.h>
#endif

#include <iostream>
#include <fstream>
#include <string>

#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/graphviz.hpp>
#include <boost/graph/graph_utility.hpp>
#include <boost/bind.hpp>

typedef boost::adjacency_list<boost::vecS, boost::vecS, boost::directedS,
                              boost::property<boost::vertex_name_t,
                                              std::string> > Graph;
typedef boost::graph_traits<Graph>::vertex_descriptor Vertex;
typedef boost::graph_traits<Graph>::vertex_iterator vertex_iter;
typedef boost::graph_traits<Graph>::edge_iterator edge_iter;

#define TIMERSEC 16
#define SQRTPI 0.5641898
#define CAMERA_Y -3.0
#define NODE_R_MAX 0.05
#define NODE_R_MIN 0.003
#define NODE_R_DIFF (NODE_R_MAX - NODE_R_MIN)
#define NODE_MAX_G 0.7
#define NODE_MIN_G 0.5
#define NODE_MAX_B 0.6
#define NODE_MIN_B 0.2
#define EDGE_MAX_G 0.5
#define EDGE_MIN_G 0.06
#define EDGE_MAX_B 0.4
#define EDGE_MIN_B 0.0
#define EDGE_MAX_ALPHA 1.0
#define EDGE_MIN_ALPHA 0.15
#define EDGE_LINES 6
#define LABEL_MAX_G 1.0
#define LABEL_MIN_G 0.4
#define LABEL_MAX_B 0.8
#define LABEL_MIN_B 0.2

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void
gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
                file, line);
        if (abort) exit(code);
    }
}
#define DISTANCE2(D, A) do {                    \
        float x2, y2, z2;                       \
        x2 = (A).x * (A).x;                     \
        y2 = (A).y * (A).y;                     \
        z2 = (A).z * (A).z;                     \
        (D) = x2 + y2 + z2;                     \
    } while (0)

#define CROSS_PRODUCT(D, A, B) do {             \
        (D).x = (A).y * (B).z - (A).z * (B).y;  \
        (D).y = (A).z * (B).x - (A).x * (B).z;  \
        (D).z = (A).x * (B).y - (A).y * (B).x;  \
    } while (0)

#define TO_SPHERICAL(D, A) do {                         \
        float tmp = (A).x * (A).x + (A).y * (A).y;      \
        float r   = sqrt(tmp + (A).z * (A).z);          \
        float rxy = sqrt(tmp);                          \
        if (r > 0.0) {                                  \
            (D).theta = acosf((A).z / r);               \
            (D).phi = acosf((A).x / rxy);               \
            if ((A).y < 0.0) {                          \
                (D).phi = 2 * M_PI - (D).phi;           \
            }                                           \
        } else {                                        \
            (D).theta = 0.0;                            \
            (D).phi   = 0.0;                            \
        }                                               \
    } while (0)

#define TO_RECTANGULAR(D, A, R) do {             \
        float sin_theta = sinf((A).theta);       \
        (D).x = (R) * sin_theta * cosf((A).phi); \
        (D).y = (R) * sin_theta * sinf((A).phi); \
        (D).z = (R) * cos((A).theta);            \
    } while (0)

#define GET_UV(U, V, A) do {                   \
        float cos_theta = cosf((A).theta);     \
        float cos_phi   = cosf((A).phi);       \
        float sin_phi   = sinf((A).phi);       \
        (U).x = - cos_theta * cos_phi;         \
        (U).y = - cos_theta * sin_phi;         \
        (U).z = sinf(A.theta);                 \
        (V).x = sin_phi;                       \
        (V).y = - cos_phi;                     \
        (V).z = 0.0;                           \
    } while (0)

#define QUATERNION_MUL(D, A, B) do {                                    \
        (D).w = (A).w * (B).w - (A).i * (B).i - (A).j * (B).j - (A).k * (B).k; \
        (D).i = (A).i * (B).w + (A).w * (B).i - (A).k * (B).j + (A).j * (B).k; \
        (D).j = (A).j * (B).w + (A).k * (B).i + (A).w * (B).j - (A).i * (B).k; \
        (D).k = (A).k * (B).w - (A).j * (B).i + (A).i * (B).j + (A).w * (B).k; \
    } while (0)

#define ROTATE(A, V, RAD) do {                  \
        rn_quaternion p, q, r;                  \
        float r2 = (RAD) * -0.5;                \
        float sin_rad2 = sinf(r2);              \
        float cos_rad2 = cosf(r2);              \
                                                \
        p.w = 0.0;                              \
        p.i = (A).x;                            \
        p.j = (A).y;                            \
        p.k = (A).z;                            \
                                                \
        q.w = cos_rad2;                         \
        q.i = (V).x * sin_rad2;                 \
        q.j = (V).y * sin_rad2;                 \
        q.k = (V).z * sin_rad2;                 \
                                                \
        r.w = cos_rad2;                         \
        r.i = - q.i;                            \
        r.j = - q.j;                            \
        r.k = - q.k;                            \
                                                \
        rn_quaternion tmp, result;              \
                                                \
        QUATERNION_MUL(tmp, r, p);              \
        QUATERNION_MUL(result, tmp, q);         \
                                                \
        (A).x = result.i;                       \
        (A).y = result.j;                       \
        (A).z = result.k;                       \
    } while (0)

#define NORMALIZE(V) do {                                               \
        float d = sqrtf((V).x * (V).x + (V).y * (V).y + (V).z * (V).z); \
        if (d > 0.0001) {                                               \
            d = 1.0 / d;                                                \
            (V).x *= d;                                                 \
            (V).y *= d;                                                 \
            (V).z *= d;                                                 \
        } else {                                                        \
            (V).x = (V).y = (V).z = 0.0;                                \
        }                                                               \
    } while (0)

#define TO_CUDA_MEM(D, ADDRCPU, HCPU, HCUDA) do {       \
            if ((ADDRCPU) == NULL) {                    \
                (D) = NULL;                             \
            } else {                                    \
                (D) = (HCUDA) + ((ADDRCPU) - (HCPU));   \
            }                                           \
        } while (0);

rinne rinne_inst;

__device__ __constant__ int num_node_cuda;
__device__ __constant__ float factor_repulse_cuda;
__device__ __constant__ float factor_step_cuda;
__device__ __constant__ float factor_spring_cuda;

__global__ void force_directed(rn_node *p_node, rn_pos *p_pos);

void
render_string(float x, float y, float z, std::string const& str)
{
    int len;
    
    glRasterPos3f(x, y, z);

    len = str.size();

    for (int i = 0; i < len; i++) {
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, str.at(i));
    }
}

void
run()
{
    int num_node = rinne_inst.get_num_node();

    if (num_node < 2 )
        return;

    int block_size;
    int min_grid_size;
    int grid_size;
    int total_thread;

    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
                                                 force_directed, 0, num_node));

    grid_size = (num_node + block_size - 1) / block_size;
    total_thread = grid_size * block_size;

    float factor_repulse = rinne_inst.get_factor_repulse();
    float factor_step = rinne_inst.get_factor_step();
    float factor_spring = rinne_inst.get_factor_spring();

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(num_node_cuda), &num_node,
                                 sizeof(num_node_cuda)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(factor_repulse_cuda), &factor_repulse,
                                 sizeof(factor_repulse_cuda)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(factor_step_cuda), &factor_step,
                                 sizeof(factor_step_cuda)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(factor_spring_cuda), &factor_spring,
                                 sizeof(factor_spring_cuda)));

    std::cout << "grid size = " << grid_size
              << "\nblock size = " << block_size
              << "\ntotal thread = " << total_thread << std::endl;

    for (int i = 0; i < 200; i++) {
        force_directed<<<grid_size, block_size>>>(rinne_inst.get_node_cuda(),
                                                  rinne_inst.get_pos_cuda());
        gpuErrchk(hipDeviceSynchronize());
        rinne_inst.copy_result();

        std::cout << i << std::endl;

        if (i == 25) {
            rinne_inst.reduce_step();
        } else if (i == 50) {
            rinne_inst.reduce_step();
        } else if (i == 100) {
            rinne_inst.reduce_step();
        }
        //usleep(100000);
    }
}

void
rinne::update_time()
{
    double  t, diff, r;
    double  cycle = m_cycle * 0.5;
    timeval tv;
    
    gettimeofday(&tv, NULL);

    t = (double)tv.tv_sec + (double)tv.tv_usec * 0.000001;

    m_prev_sec = m_current_sec;
    m_current_sec = t;

    diff = t - m_init_sec;

    r = sin(M_PI * (diff - M_PI * 0.5) / cycle) * 0.5 + 0.5;

    if (diff > cycle && r < 0.004) {
        m_init_sec = t;
        m_top_idx++;
        if (m_top_idx > m_top_n)
            m_top_idx = 0;
    }
}

void
rinne::rotate_view()
{
    if (m_is_auto_rotate && m_top_idx  > 0) {
        float theta = m_node_top[m_top_idx - 1]->pos.theta + M_PI_2;
        float diff, tmp;

        theta += m_rotate_x * 2 * M_PI;

        if (theta > 2 * M_PI) {
            theta -= 2 * M_PI;
        }

        diff = M_PI - theta;

        if (fabs(diff) > 0.05) {
            if (diff > 0.0) {
                m_rotate_x += (m_current_sec - m_prev_sec) / (m_cycle * 4.0);
            } else {
                m_rotate_x -= (m_current_sec - m_prev_sec) / (m_cycle * 4.0);
            }

            m_rotate_x = modf(m_rotate_x, &tmp);
            if (m_rotate_x < 0.0) {
                m_rotate_x += 1.0;
            }
        }

        float phi = m_node_top[m_top_idx - 1]->pos.phi;

        phi += m_rotate_z * 2 * M_PI;

        if (phi > 2 * M_PI) {
            phi -= 2 * M_PI;
        }

        diff = 3 * M_PI_2 - phi;
        if (fabs(diff) > 0.05) {
            if (0 < diff && diff < M_PI) {
                m_rotate_z += (m_current_sec - m_prev_sec) / (m_cycle * 4.0);
            } else {
                m_rotate_z -= (m_current_sec - m_prev_sec) / (m_cycle * 4.0);
            }

            m_rotate_z = modf(m_rotate_z, &tmp);
            if (m_rotate_z < 0.0) {
                m_rotate_z += 1.0;
            }
        }
    }
}

void
rinne::display()
{
    update_time();
    rotate_view();

    glClearColor(0.0, 0.0, 0.0, 1.0);
    glClear(GL_COLOR_BUFFER_BIT);
    glFlush();

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(40.0, (double)m_window_w / (double)m_window_h, 0.1, 8);
    gluLookAt(0.0, CAMERA_Y, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0);
    glMatrixMode(GL_MODELVIEW);

    glPushMatrix();

    glRotatef(360 * m_rotate_x, 1.0, 0.0, 0.0);
    glRotatef(360 * m_rotate_z, 0.0, 0.0, 1.0);

    glColor3f(0.4, 0.4, 0.4);
    glutWireSphere(1.0, 16, 16);

    draw_node();
    //draw_tau();

    glPopMatrix();

    glutSwapBuffers();
}

void
display()
{
    rinne_inst.display();
}

void
on_keyboard(unsigned char key, int x, int y)
{
    rinne_inst.on_keyboard(key, x, y);
}

void
rinne::on_keyboard(unsigned char key, int x, int y)
{
    switch (key) {
    case 'f':
    case 'F':
        if (m_is_fullscreen) {
            glutPositionWindow(0,0);
            glutReshapeWindow(1200, 900);
        } else {
            glutFullScreen();
        }
        m_is_fullscreen = ! m_is_fullscreen;
        break;
    case 'q':
    case 'Q':
    case '\033': // ESC
        exit(0);
    default:
        break;
    }
}

void
on_resize(int w, int h)
{
    rinne_inst.on_resize(w, h);
}

void
on_mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN) {
        rinne_inst.on_mouse_down(button, x, y);
    } else if (state == GLUT_UP) {
        rinne_inst.on_mouse_up(button, x, y);
    }
}

void
on_mouse_move(int x, int y)
{
    rinne_inst.on_mouse_move(x, y);
}

void
glut_timer(int val)
{
    glutPostRedisplay();
    glutTimerFunc(TIMERSEC, glut_timer, 0);
}

void
init_glut(int argc, char *argv[])
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowSize(1024, 768);
    glutCreateWindow(argv[0]);
    glutDisplayFunc(display);
    glutKeyboardFunc(on_keyboard);
    glutReshapeFunc(on_resize);
    glutMouseFunc(on_mouse);
    glutMotionFunc(on_mouse_move);
    glutTimerFunc(TIMERSEC, glut_timer, 0);

    glEnable(GL_BLEND);
    glBlendFunc(GL_ONE, GL_ONE_MINUS_SRC_ALPHA);
    glBlendEquation(GL_FUNC_ADD);

    glutMainLoop();
}

void
rinne::on_resize(int w, int h)
{
    glViewport(0, 0, w, h);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(40.0, (double)w / (double)h, 0.1, 8);
    gluLookAt(0.0, CAMERA_Y, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0);
    glMatrixMode(GL_MODELVIEW);

    m_window_w = w;
    m_window_h = h;
}

void
rinne::on_mouse_down(int button, int x, int y)
{
    m_is_mouse_down = true;
    m_mouse_x = x;
    m_mouse_y = y;
}

void
rinne::on_mouse_up(int button, int x, int y)
{
    m_is_mouse_down = false;
}

void
rinne::on_mouse_move(int x, int y)
{
    if (m_is_mouse_down && ! m_is_auto_rotate) {
        int dx = x - m_mouse_x;
        int dy = y - m_mouse_y;

        m_rotate_z += dx * 0.001;
        m_rotate_x += dy * 0.001;

        double tmp;
        m_rotate_z = modf(m_rotate_z, &tmp);
        if (m_rotate_z < 0.0) {
            m_rotate_z += 1.0;
        }

        m_rotate_x = modf(m_rotate_x, &tmp);
        if (m_rotate_x < 0.0) {
            m_rotate_x += 1.0;
        }

        m_mouse_x = x;
        m_mouse_y = y;

        glutPostRedisplay();
    }
}

void
rinne::draw_label()
{
    double g, b, alpha;
    rn_vec v;

    for (int i = 0; i < m_top_n; i++) {
        if (m_is_blink && i + 1 == m_top_idx)
            continue;

        TO_RECTANGULAR(v, m_node_top[i]->pos, 1.0);

        glColor3f(0.0, 0.0, 0.0);
        render_string(v.x - 0.005, v.y, v.z - 0.005,
                      m_label[(m_node_top[i] - m_node)]);

        get_color(g, b, alpha, LABEL_MIN_G, LABEL_MAX_G,
                  LABEL_MIN_B, LABEL_MAX_B, 0.0, 0.0);

        if (m_is_blink) {
            if (m_top_idx == 0) {
                glColor3f(0.0, g, b);
            } else{
                glColor3f(0.0, LABEL_MIN_G, LABEL_MIN_B);
            }
        } else {
            glColor3f(0.0, LABEL_MAX_G, LABEL_MAX_B);
        }

        render_string(v.x, v.y, v.z, m_label[(m_node_top[i] - m_node)]);
    }

    if (m_is_blink && m_top_idx > 0) {
        TO_RECTANGULAR(v, m_node_top[m_top_idx - 1]->pos, 1.0);
        
        glColor3f(0.0, 0.0, 0.0);
        render_string(v.x - 0.005, v.y, v.z - 0.005,
                      m_label[(m_node_top[m_top_idx - 1] - m_node)]);

        get_color(g, b, alpha, LABEL_MIN_G, LABEL_MAX_G,
                  LABEL_MIN_B, LABEL_MAX_B, 0.0, 0.0);
        glColor3f(0.0, g, b);
        render_string(v.x, v.y, v.z,
                      m_label[(m_node_top[m_top_idx - 1] - m_node)]);
    }
}

void
rinne::draw_edge(double g, double b, double alpha)
{
    rn_node *dst = m_node_top[m_top_idx - 1];

    for (rn_node *p = m_node; p != &m_node[m_num_node]; p++) {
        rn_vec ev, ev1, ev2;
        double cos_theta_a = cos(p->pos.theta);
        double sin_theta_a = sin(p->pos.theta);

        TO_RECTANGULAR(ev, p->pos, 1.0);

        for (rn_edge *p_edge = p->edge; p_edge != NULL; p_edge = p_edge->next) {
            rn_pos delta;

            delta.theta = p->pos.theta - p_edge->dst->pos.theta;
            delta.phi   = p->pos.phi - p_edge->dst->pos.phi;

            if (delta.phi > M_PI) {
                delta.phi = -(2 * M_PI - delta.phi);
            } else if (delta.phi < -M_PI) {
                delta.phi = 2 * M_PI + delta.phi;
            }
            
            if (!m_is_blink || m_top_idx == 0 || dst == p_edge->dst) {
                glColor4f(0.0, g, b, alpha);
            } else {
                glColor4f(0.0, EDGE_MIN_G, EDGE_MIN_B, EDGE_MIN_ALPHA);
            }

            if (m_is_blink && m_top_idx != 0 && dst == p_edge->dst) {
                glLineWidth(2.0f);
            } else {
                glLineWidth(1.0f);
            }

            TO_RECTANGULAR(ev2, p_edge->dst->pos, 1.0);

            double psi = acos(cos_theta_a * cos(p_edge->dst->pos.theta) +
                              sin_theta_a * sin(p_edge->dst->pos.theta) *
                              cos(p->pos.phi - p_edge->dst->pos.phi));

            rn_vec cross;

            ev1 = ev;
            psi /= EDGE_LINES;

            CROSS_PRODUCT(cross, ev, ev2);
            NORMALIZE(cross);
            glBegin(GL_LINE_STRIP);

            glVertex3f(ev.x, ev.y , ev.z);

            for (int i = 0; i < EDGE_LINES - 1; i++) {
                ROTATE(ev1, cross, psi);
                glVertex3f(ev1.x, ev1.y, ev1.z);
            }

            glVertex3f(ev2.x, ev2.y , ev2.z);

            glEnd();
            glLineWidth(1.0f);
        }
    }
}

void
rinne::draw_node()
{
    double r_denom = 1.0 / m_max_in_degree;
    double max_alpha = 1.0;
    double min_alpha = 0.0;
    double max_g;
    double min_g;
    double max_b;
    double min_b;
    double b, g, alpha;

    // draw far side nodes
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(40.0, (double)m_window_w / (double)m_window_h, -CAMERA_Y, 8);
    gluLookAt(0.0, CAMERA_Y, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0);
    glMatrixMode(GL_MODELVIEW);

    glColor3d(0.0, NODE_MIN_G * 0.125, NODE_MIN_B * 0.125);
    for (rn_node *p = m_node; p != &m_node[m_num_node]; p++) {
        rn_vec a;
        double r = p->num_bp_edge * r_denom;

        r = r * NODE_R_DIFF + NODE_R_MIN;

        TO_RECTANGULAR(a, p->pos, 1.0);

        glPushMatrix();

        glTranslatef(a.x, a.y, a.z);
        glutSolidSphere(r, 8, 8);

        glPopMatrix();
    }

    // draw edges
    if (m_is_blink) {
        max_alpha = EDGE_MAX_ALPHA;
        min_alpha = EDGE_MIN_ALPHA;
        max_g = EDGE_MAX_G;
        min_g = EDGE_MIN_G;
        max_b = EDGE_MAX_B;
        min_b = EDGE_MIN_B;

        get_color(g, b, alpha, min_g, max_g, min_b, max_b, min_alpha, max_alpha);
    } else {
        g = EDGE_MAX_G;
        b = EDGE_MAX_B;
        alpha = EDGE_MAX_ALPHA;
    }

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(40.0, (double)m_window_w / (double)m_window_h,
                   0.1, -CAMERA_Y);
    gluLookAt(0.0, CAMERA_Y, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0);
    glMatrixMode(GL_MODELVIEW);
    draw_edge(g, b, alpha);


    // draw near side nodes
    if (m_is_blink) {
        max_g = NODE_MAX_G;
        min_g = NODE_MIN_G;
        max_b = NODE_MAX_B;
        min_b = NODE_MIN_B;

        get_color(g, b, alpha, min_g, max_g, min_b, max_b, min_alpha, max_alpha);
    } else {
        g = NODE_MAX_G;
        b = NODE_MIN_G;
    }

    rn_node *dst = m_node_top[m_top_idx - 1];

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(40.0, (double)m_window_w / (double)m_window_h,
                   0.1, -CAMERA_Y);
    gluLookAt(0.0, CAMERA_Y, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0);
    glMatrixMode(GL_MODELVIEW);

    for (rn_node *p = m_node; p != &m_node[m_num_node]; p++) {
        rn_vec a;
        double r = p->num_bp_edge * r_denom;

        r = r * NODE_R_DIFF + NODE_R_MIN;

        TO_RECTANGULAR(a, p->pos, 1.0);

        if (!m_is_blink || m_top_idx == 0 || p == dst) {
            glColor3f(0.0, g, b);
        } else {
            glColor3f(0.0, NODE_MIN_G, NODE_MIN_B);
        }

        glPushMatrix();

        glTranslatef(a.x, a.y, a.z);
        glutSolidSphere(r, 8, 8);

        glPopMatrix();
    }

    draw_label();
}

__device__
void
get_spring_vec(rn_vec &uv, float psi)
{
    float power;
    float p = psi / M_PI + 1.0f;

    p *= p;
    p *= p;
    p *= p;
    p *= p;
    p *= p;

    power = factor_step_cuda * factor_spring_cuda * p;

    uv.x *= power;
    uv.y *= power;
    uv.z *= power;
}

__device__
void
get_repulse_vec(rn_vec &uv, float psi)
{
    float power;
    float p = psi + M_PI;

    power = - factor_step_cuda * factor_repulse_cuda / (p * p);

    uv.x *= power;
    uv.y *= power;
    uv.z *= power;
}

__device__
void
get_uv_vec(rn_vec &v, const rn_pos &a, const rn_pos &b)
{
    rn_vec va, vb;
    float t;

    TO_RECTANGULAR(va, a, 1.0);
    TO_RECTANGULAR(vb, b, 1.0);

    t = 1.0f - va.x * vb.x - va.y * vb.y - va.z * vb.z;

    v.x = vb.x + va.x * t;
    v.y = vb.y + va.y * t;
    v.z = vb.z + va.z * t;

    v.x -= va.x;
    v.y -= va.y;
    v.z -= va.z;

    NORMALIZE(v);
}

int
cmp_node(const void *lhs, const void *rhs)
{
    const rn_node *p1 = *(rn_node**)lhs;
    const rn_node *p2 = *(rn_node**)rhs;

    if (p1->num_bp_edge > p2->num_bp_edge)
        return -1;

    if (p1->num_bp_edge < p2->num_bp_edge)
        return 1;

    return 0;
}

void
rinne::get_top_n()
{
    rn_node **p;

    if (m_num_node < m_top_n)
        m_top_n = m_num_node;

    p = new rn_node*[m_num_node];
    m_node_top = new rn_node*[m_top_n];

    for (int i = 0; i < m_num_node; i++) {
        p[i] = &m_node[i];
    }

    qsort(p, m_num_node, sizeof(p), cmp_node);

    for (int j = 0; j < m_top_n; j++) {
        m_node_top[j] = p[j];
    }

    delete[] p;
}

void
rinne::copy_result()
{
    rn_pos  *p_pos = new rn_pos[m_num_node];
    rn_node *p_node = new rn_node[m_num_node];
    
    gpuErrchk(hipMemcpy(p_pos, m_pos_cuda, sizeof(*p_pos) * m_num_node,
                         hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(p_node, m_node_cuda, sizeof(*p_node) * m_num_node,
                         hipMemcpyDeviceToHost));

    for (int i = 0; i < m_num_node; i++) {
        p_node[i].pos = m_node[i].pos = p_pos[i];
    }
    
    gpuErrchk(hipMemcpy(m_node_cuda, p_node, sizeof(*p_node) * m_num_node,
                         hipMemcpyHostToDevice));

    delete[] p_pos;
    delete[] p_node;
}

void
rinne::get_uv_vec_rand(rn_vec &v, const rn_pos &a)
{
    static int i = 0, j = 0;
    static float theta[7] = {0.0,
                             M_PI / 7.0,
                             2 * M_PI / 7.0,
                             3 * M_PI / 7.0,
                             4 * M_PI / 7.0,
                             5 * M_PI / 7.0,
                             6 * M_PI / 7.0};
    static float phi[11] = {0.0,
                            M_PI / 11.0,
                            2 * M_PI / 11.0,
                            3 * M_PI / 11.0,
                            4 * M_PI / 11.0,
                            5 * M_PI / 11.0,
                            6 * M_PI / 11.0,
                            7 * M_PI / 11.0,
                            8 * M_PI / 11.0,
                            9 * M_PI / 11.0,
                            10 * M_PI / 11.0};

    for (;;) {
        rn_pos b;
        b.theta = theta[i++];
        b.phi   = phi[j++];

        if (i >= 7)
            i = i % 7;
        if (j >= 11)
            j = j % 11;

        float psi;
        psi = acosf(cosf(a.theta) * cosf(b.theta) +
                    sinf(a.theta) * sinf(b.theta) *
                    cosf(a.phi - b.phi));

        if (isnan(psi))
            continue;

        //get_uv_vec(v, a, b);
        //get_repulse_vec(v, 0.0001);

        break;
    }
}

__global__
void
force_directed(rn_node *p_node, rn_pos *p_pos)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    rn_node *p1;
    p_pos = &p_pos[id];

    //    if (id < num_node_cuda) {
    for (p1 = &p_node[id]; p1 < p_node + num_node_cuda; p1 += num_node_cuda) {
        rn_node *p2;
        rn_vec v1 = {0.0, 0.0, 0.0};
        rn_vec v2;
        float cos_theta_a, sin_theta_a;
        float psi;

        cos_theta_a = cosf(p1->pos.theta);
        sin_theta_a = sinf(p1->pos.theta);

        for (p2 = p_node; p2 != &p_node[num_node_cuda]; p2++) {
            if (p1 == p2)
                continue;

            psi = acosf(cos_theta_a * cosf(p2->pos.theta) +
                        sin_theta_a * sinf(p2->pos.theta) *
                        cosf(p1->pos.phi - p2->pos.phi));

            get_uv_vec(v2, p1->pos, p2->pos);
            get_repulse_vec(v2, psi);

            v1.x += v2.x;
            v1.y += v2.y;
            v1.z += v2.z;
        }

        rn_edge *p_edge;
        for (p_edge = p1->edge; p_edge != NULL; p_edge = p_edge->next) {
            rn_vec v3;

            psi = acosf(cos_theta_a * cosf(p_edge->dst->pos.theta) +
                        sin_theta_a * sinf(p_edge->dst->pos.theta) *
                        cosf(p1->pos.phi - p_edge->dst->pos.phi));

            if (isnan(psi))
                continue;

            get_uv_vec(v3, p1->pos, p_edge->dst->pos);
            get_spring_vec(v3, psi);

            v1.x += v3.x;
            v1.y += v3.y;
            v1.z += v3.z;
        }

        for (p_edge = p1->bp_edge; p_edge != NULL; p_edge = p_edge->bp_next) {
            rn_vec v3;

            psi = acosf(cos_theta_a * cosf(p_edge->src->pos.theta) +
                        sin_theta_a * sinf(p_edge->src->pos.theta) *
                        cosf(p1->pos.phi - p_edge->src->pos.phi));

            if (isnan(psi))
                continue;

            get_uv_vec(v3, p1->pos, p_edge->src->pos);
            get_spring_vec(v3, psi);

            v1.x += v3.x;
            v1.y += v3.y;
            v1.z += v3.z;
        }

        rn_vec pvec, cross;
        double rad;

        TO_RECTANGULAR(pvec, p1->pos, 1.0);
        DISTANCE2(rad, v1);

        rad = sqrt(rad);
        if (rad > M_PI_4)
            rad = M_PI_4;

        CROSS_PRODUCT(cross, pvec, v1);
        NORMALIZE(cross);
        ROTATE(pvec, cross, rad);

        rn_pos pos = p1->pos;
        TO_SPHERICAL(pos, pvec);
        *p_pos = pos;
        p_pos += num_node_cuda;
    }
}

void
rinne::init_pos()
{
    srand(time(NULL));
    for (rn_node *p = m_node; p != &m_node[m_num_node]; p++) {
        p->pos.theta = M_PI_2 * ((double)rand() / RAND_MAX) + M_PI_4;
        p->pos.phi   = 2 * M_PI * ((double)rand() / RAND_MAX);
    }
}

void
rinne::init_graph_cuda()
{
    int i;
    rn_node *p_node = new rn_node[m_num_node];
    rn_edge *p_edge = new rn_edge[m_num_edge];

    gpuErrchk(hipMalloc((void**)&m_node_cuda,
                         sizeof(*m_node_cuda) * m_num_node));
    gpuErrchk(hipMalloc((void**)&m_edge_cuda,
                         sizeof(*m_edge_cuda) * m_num_edge));
    gpuErrchk(hipMalloc((void**)&m_pos_cuda,
                         sizeof(*m_pos_cuda) * m_num_node));

    for (i = 0; i < m_num_node; i++) {
        p_node[i].pos = m_node[i].pos;
        p_node[i].num_edge = m_node[i].num_edge;
        p_node[i].num_bp_edge = m_node[i].num_bp_edge;

        TO_CUDA_MEM(p_node[i].edge, m_node[i].edge, m_edge, m_edge_cuda);
        TO_CUDA_MEM(p_node[i].bp_edge, m_node[i].bp_edge, m_edge, m_edge_cuda);
    }

    for (i = 0; i < m_num_edge; i++) {
        TO_CUDA_MEM(p_edge[i].src, m_edge[i].src, m_node, m_node_cuda);
        TO_CUDA_MEM(p_edge[i].dst, m_edge[i].dst, m_node, m_node_cuda);
        TO_CUDA_MEM(p_edge[i].next, m_edge[i].next, m_edge, m_edge_cuda);
        TO_CUDA_MEM(p_edge[i].bp_next, m_edge[i].bp_next, m_edge, m_edge_cuda);
    }

    gpuErrchk(hipMemcpy(m_node_cuda, p_node, sizeof(*p_node) * m_num_node,
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(m_edge_cuda, p_edge, sizeof(*p_edge) * m_num_edge,
                         hipMemcpyHostToDevice));

    delete[] p_node;
    delete[] p_edge;
}

void
rinne::read_dot(char *path)
{
    std::ifstream file(path);

    Graph g;
    boost::dynamic_properties dp(boost::ignore_other_properties);
    dp.property("vertex_name", get(boost::vertex_name, g));

    boost::read_graphviz(file, g, dp, "vertex_name");

    m_num_node = num_vertices(g);
    m_num_edge = num_edges(g);

    m_node  = new rn_node[m_num_node];
    m_edge  = new rn_edge[m_num_edge];
    m_label = new std::string[m_num_node];

/*
    gpuErrchk(hipMallocManaged((void**)&m_node,
                                sizeof(*m_node) * m_num_node,
                                hipMemAttachGlobal));
    gpuErrchk(hipMallocManaged((void**)&m_edge,
                                sizeof(*m_edge) * m_num_edge,
                                hipMemAttachGlobal));
*/

    memset(m_node, 0, sizeof(*m_node) * m_num_node);
    memset(m_edge, 0, sizeof(*m_edge) * m_num_edge);
    
    std::pair<vertex_iter, vertex_iter> vp;
    int i = 0;
    for (vp = vertices(g); vp.first != vp.second; ++vp.first) {
        m_label[i++] = get(boost::vertex_name, g, *vp.first);
    }

    std::pair<edge_iter, edge_iter> ep;
    rn_edge *p_edge = m_edge;
    for (ep = edges(g); ep.first != ep.second; ++ep.first) {
        int s = source(*ep.first, g);
        int t = target(*ep.first, g);

        p_edge->src = &m_node[s];
        p_edge->dst = &m_node[t];

        p_edge->next = p_edge->src->edge;
        p_edge->src->edge = p_edge;
        p_edge->bp_next = p_edge->dst->bp_edge;
        p_edge->dst->bp_edge = p_edge;
        p_edge->src->num_edge++;
        p_edge->dst->num_bp_edge++;

        if (m_max_out_degree < p_edge->src->num_edge)
            m_max_out_degree = p_edge->src->num_edge;

        if (m_max_in_degree < p_edge->dst->num_bp_edge)
            m_max_in_degree = p_edge->dst->num_bp_edge;

        p_edge++;
    }

    init_pos();

    std::cout << "#node = " << m_num_node << std::endl;
    std::cout << "#edge = " << m_num_edge << std::endl;

    m_factor_step /= m_num_node * 100;

    get_top_n();
    init_graph_cuda();

    run();
}

void
rinne::get_color(double &g, double &b, double &alpha,
                 double min_g, double max_g,
                 double min_b, double max_b,
                 double min_alpha, double max_alpha)
{
    double diff_g = max_g - min_g;
    double diff_b = max_b - min_b;
    double diff_alpha = max_alpha - min_alpha;
    double diff, r, cycle;

    cycle = m_cycle * 0.5;

    diff = m_current_sec - m_init_sec;

    r = sin(M_PI * (diff - M_PI * 0.5) / cycle) * 0.5 + 0.5;

    //std::cout << r << std::endl;

    g = r * diff_g + min_g;
    b = r * diff_b + min_b;
    alpha = r * diff_alpha + min_alpha;
}

int
main(int argc, char *argv[])
{
    if (argc < 2) {
        std::cerr << "usage: " << argv[0] << " graph.dot" << std::endl;
        return 1;
    }

    std::cout << "loading " << argv[1] << " ..." << std::endl;
    rinne_inst.read_dot(argv[1]);

    init_glut(argc, argv);

    return 0;
}
